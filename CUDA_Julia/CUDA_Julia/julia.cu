#include "hip/hip_runtime.h"
#include "common/book.h"
#include "common/cpu_bitmap.h"

#define DIM 1000

__device__ int julia(int x, int y)
{
	const float scale = 1.5;
	float jx = scale * (float)(DIM/2 - x) / (DIM/2);
	float jy = scale * (float)(DIM/2 - y) / (DIM/2);

	float c_r = -0.8, c_i = 0.156;
	float a_r = jx, a_i = jy;

	for (int i = 0; i < 200; i++)
	{
		float next_a_r = a_r * a_r - a_i * a_i + c_r;
		float next_a_i = a_i * a_r + a_r * a_i + c_i;

		a_r = next_a_r; 
		a_i = next_a_i;

		if (a_r * a_r + a_i * a_i > 1000)
			return 0;
	}

	return 1;
}

__global__ void kernel(unsigned char *ptr)
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	int juliaValue = julia(x, y);
	ptr[offset * 4 + 0] = 255 * juliaValue;
	ptr[offset * 4 + 1] = 0;
	ptr[offset * 4 + 2] = 0;
	ptr[offset * 4 + 3] = 255;
}

int main() 
{
	CPUBitmap bitmap(DIM, DIM, NULL); 
	unsigned char *device_bitmap;

	HANDLE_ERROR( hipMalloc((void**)&device_bitmap, bitmap.image_size()) );

	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(device_bitmap);

	HANDLE_ERROR( hipMemcpy(bitmap.get_ptr(), device_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost) );

	bitmap.display_and_exit(NULL);

	HANDLE_ERROR( hipFree(device_bitmap) );

	return 0;
}
